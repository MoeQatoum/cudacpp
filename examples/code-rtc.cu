
#include <hip/hip_runtime.h>

//#include "cudacpp\DeviceVector.h"

template<typename type, int size>
__global__ void setKernel(type* c, type val)
{
	auto idx = threadIdx.x * size;

	#pragma unroll(size)
	for (auto i = 0; i < size; i++) {
		c[idx] = val;
		idx++;
	}
}