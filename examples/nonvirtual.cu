#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include <stdio.h>

#include "cudacpp\DeviceVector.h"


namespace nonvirtual {

class SimpleIntProvider
{
	int _i;

public:
	__device__ __inline__ SimpleIntProvider(int i) : _i(i) {}
	__device__ __inline__ auto getNumber() const { return _i; }
};


class SumIntProvider
{
	int _a;
	int _b;

public:
	__device__ __inline__ SumIntProvider(int a, int b) : _a(a), _b(b) {}
	__device__ __inline__ auto getNumber() const { return _a + _b; }

};


template<typename T>
__device__ __inline__ void putValue(int& to, const T& ip) {
	to = ip.getNumber();
}



__global__ void addKernel(cudacpp::DeviceVector<int> c, int val)
{
	auto idx = threadIdx.x;
	//SimpleIntProvider sip{ val };
	SumIntProvider sip{ val, 2 };
	putValue(c[idx], sip);
}

__global__ void addKernelDirect(cudacpp::DeviceVector<int> c, int val)
{
	auto idx = threadIdx.x;
	//SimpleIntProvider sip{ val };
	//SumIntProvider sip{ val, 2 };
	c[idx] = val + 2;
}

}

int testNonVirtual(int size, int *c, int val) {
	hipError_t cudaStatus;


	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return cudaStatus;
	}

	// Allocate GPU buffers for three vectors (two input, one output) 
	cudacpp::DeviceMemoryT<int> dev_c(size);

	// Launch a kernel on the GPU with one thread for each element.
	nonvirtual::addKernel << <1, size >> >(dev_c, val);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		return cudaStatus;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return cudaStatus;
	}

	return cudaStatus;


	return 0;
}