#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "cudacpp\DeviceVector.h"
#include "cudacpp\Index.h"
#include "cudacpp\Grid.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

template<int DIM_BLOCKS, int DIM_THREADS, typename T>
__global__ void addKernel(cudacpp::DeviceVector<T> c, const cudacpp::DeviceVector<T> a, const cudacpp::DeviceVector<T> b)
{
	auto idx = cudacpp::Index<1>::create<DIM_BLOCKS, DIM_THREADS>();
	if (idx.inRange(c.size())) {
		c[idx] = a[idx] + b[idx];
	}
}

int basics(int size, const int *a, const int *b, int *c)
{
    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return cudaStatus;
    }

    // Allocate GPU buffers for three vectors (two input, one output) 
	cudacpp::DeviceMemory dev_a(sizeof(int) * size);
	cudacpp::DeviceMemoryT<int> dev_b(size);
	cudacpp::DeviceMemoryT<int> dev_c(size);

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
		return cudaStatus;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
		return cudaStatus;
    }

    // Launch a kernel on the GPU with one thread for each element.

	cudacpp::DeviceVector<int> vec_a{ dev_a, size };

	auto grid = cudacpp::CreateGrid(cudacpp::Size<1>{4}, vec_a.size());

    addKernel<grid.DimBlocks, grid.DimThreads, int><<<grid.blocks, grid.threads>>>(dev_c, vec_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		return cudaStatus;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
		return cudaStatus;
    }

    return cudaStatus;
}
